#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <math.h>
#include <boost/thread.hpp>


__global__ void myKernel(void)
{

}
__global__ void add(float* a, float* b, float* c, int n)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i < n)
        c[i] = a[i] + b[i];
}
#define SIZE 10000000

int main(void)
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    h_a = new float[SIZE];
    h_b = new float[SIZE];
    h_c = new float[SIZE];
    if(h_a == NULL || h_b == NULL || h_c == NULL) return -1;
    size_t size = SIZE*sizeof(float);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    for(int i = 0; i < SIZE; ++i)
    {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);

    int blockSize = 1024;
    int gridSize = (int)ceil((float)SIZE/blockSize);
    boost::posix_time::ptime start = boost::posix_time::microsec_clock::universal_time();
    add<<<gridSize,blockSize>>>(d_a,d_b,d_c,SIZE);
    boost::posix_time::ptime end = boost::posix_time::microsec_clock::universal_time();
    boost::posix_time::time_duration delta = end - start;
    std::cout << "Calculation on the GPU took: " << delta.total_microseconds() << std::endl;
    hipMemcpy(h_c, d_c, size,hipMemcpyDeviceToHost);
    start = boost::posix_time::microsec_clock::universal_time();
    for(int i = 0; i < SIZE; ++i)
    {
        if(fabs(h_a[i] + h_b[i] - h_c[i]) > 1e-5)
        {
            std::cout << "error " << i << std::endl;
        }
    }
    end = boost::posix_time::microsec_clock::universal_time();
    delta = end - start;
    std::cout << "Calculation on the CPU took: " << delta.total_microseconds() << std::endl;
    std::cout<< "Hello World!\n";
    return 0;
}
